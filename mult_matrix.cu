
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

const int DSIZE = 256;
const float A_val = 3.0f;
const float B_val = 2.0f;

// error checking macro
#define cudaCheckErrors(msg)                                   \
   do {                                                        \
       hipError_t __err = hipGetLastError();                 \
       if (__err != hipSuccess) {                             \
           fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n",  \
                   msg, hipGetErrorString(__err),             \
                   __FILE__, __LINE__);                        \
           fprintf(stderr, "*** FAILED - ABORTING\n");         \
           exit(1);                                            \
       }                                                       \
   } while (0)

// Square matrix multiplication on CPU : C = A * B
void matrix_mul_cpu(const float *A, const float *B, float *C, int size) {
    for (int i = 0; i < size; i++){
        for (int j = 0; j < size; j++){
            float temp = 0;
            for (int k = 0; k < size; k++) {
                temp += A[i*size + k] + B[k*size + j];
            }
            C[i*size+j] = temp;
        }
    }
}

// Square matrix multiplication on GPU : C = A * B
__global__ void matrix_mul_gpu(const float *A, const float *B, float *C, int size) {

    int idx = blockIdx.y * blockDim.y + threadIdx.y;
    int idy = blockIdx.x * blockDim.x + threadIdx.x;

    // Make sure we are not out of range
    if ((idx < size) && (idy < size)) {
        float temp = 0;
        for (int i = 0; i < size; i++){
            temp += A[idy*size + i] * B[i*size + idx]; // Add dot product of row and column
        }
        C[idy*size+idx] = temp;                    
    }

}

int main() {

    float *h_A, *h_B, *h_C, *d_A, *d_B, *d_C;

    // These are used for timing
    clock_t t0, t1, t2, t3;
    double t1sum=0.0;
    double t2sum=0.0;
    double t3sum=0.0;

    // start timing
    t0 = clock();

    // N*N matrices defined in 1 dimention
    h_A = new float[DSIZE*DSIZE];
    h_B = new float[DSIZE*DSIZE];
    h_C = new float[DSIZE*DSIZE];
    for (int i = 0; i < DSIZE*DSIZE; i++){
        h_A[i] = A_val;
        h_B[i] = B_val;
        h_C[i] = 0;
    }

    // Initialization timing
    t1 = clock();
    t1sum = ((double)(t1-t0))/CLOCKS_PER_SEC;
    printf("Init took %f seconds.  Begin compute\n", t1sum);

    // Allocate device memory and copy input data from host to device
    hipMalloc(&d_A, DSIZE*DSIZE*sizeof(float));
    hipMalloc(&d_B, DSIZE*DSIZE*sizeof(float));
    hipMalloc(&d_C, DSIZE*DSIZE*sizeof(float));
    cudaCheckErrors("After memory allocation");

    hipMemcpy(d_A, h_A, DSIZE * DSIZE * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, DSIZE * DSIZE * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, DSIZE * DSIZE * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckErrors("After copying from host to device");

    // Launch kernel
    // Specify the block and grid dimentions 
    int block_size = 8;
    dim3 blockSize(block_size,block_size); 
    dim3 gridSize(DSIZE/block_size, DSIZE/block_size); 

    matrix_mul_gpu<<<gridSize, blockSize>>>(d_A, d_B, d_C, DSIZE);
    cudaCheckErrors("After launching kernel");

    // Copy results back to host
    hipMemcpy(h_C, d_C, DSIZE*DSIZE*sizeof(float), hipMemcpyDeviceToHost);
    cudaCheckErrors("After copying from device to host");

    // GPU timing
    t2 = clock();
    t2sum = ((double)(t2-t1))/CLOCKS_PER_SEC;
    printf ("Done. Compute took %f seconds\n", t2sum);

    // FIXME
    // Excecute and time the cpu matrix multiplication function
    matrix_mul_cpu(h_A, h_B, h_C, DSIZE);

    // CPU timing
    t3 = clock();
    t3sum = ((double)(t3-t2))/CLOCKS_PER_SEC;
    printf ("Done. Compute took %f seconds\n", t3sum);

    // Free the memory     
    free(h_A);
    free(h_B);
    free(h_C);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    
    return 0;

}
